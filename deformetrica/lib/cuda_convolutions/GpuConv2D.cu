#include "hip/hip_runtime.h"
#ifndef _GpuConv2D_cu
#define _GpuConv2D_cu

#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "GaussFunction.h"
#include "ScalarRadialKernel.h"

template <typename TYPE, int DIMVECT>
__global__ void reduce0(TYPE* in, TYPE* out, int sizeY,int nx)
{
	TYPE res = 0;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < nx*DIMVECT)
    {
		for (int i = 0; i < sizeY; i++)
            res += in[tid + i*nx*DIMVECT];
		/*res = in[tid+ nx* DIMVECT];*/
		out[tid] = res;
	}
}


///////////////////////////////////////
///// Conv2D ////////////////////////////
///////////////////////////////////////


// thread kernel: computation of gammai = sum_j k(xi,yj)betaj for index i given by thread id.
template < typename TYPE, int DIMPOINT, int DIMVECT, class KER  >
__global__ void GpuConv2DOnDevice(KER Ker,
                                      TYPE *x, TYPE *y, TYPE *beta, TYPE *gammaB,
                                      int nx, int ny)
{
    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    TYPE xi[DIMPOINT], gammai[DIMVECT], *yj, *betaj;
    if(i<nx)  // we will compute gammai only if i is in the range
    {
        // load xi from device global memory
        for(int k=0; k<DIMPOINT; k++)
            xi[k] = x[i*DIMPOINT+k];
        for(int k=0; k<DIMVECT; k++)
            gammai[k] = 0.0f;
    }

    int j = blockIdx.y * blockDim.x + threadIdx.x;
    int inc = DIMPOINT + DIMVECT;
    if(j<ny) // we load yj and betaj from device global memory only if j<ny
    {
        for(int k=0; k<DIMPOINT; k++)
            SharedData[threadIdx.x*inc+k] = y[j*DIMPOINT+k];
        for(int k=0; k<DIMVECT; k++)
            SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
    }
    __syncthreads();

    if(i<nx) // we compute gammai only if needed
    {
        yj = SharedData;
        betaj = SharedData + DIMPOINT;
        int inc = DIMPOINT + DIMVECT;
        for(int jrel = 0; (jrel<blockDim.x) && ((blockDim.x*blockIdx.y+jrel)< ny); jrel++, yj+=inc, betaj+=inc)
            Ker.Eval(gammai,xi,yj,betaj);
        __syncthreads();
    }

    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMVECT; k++)
            gammaB[blockIdx.y*DIMVECT*nx+i*DIMVECT+k] = gammai[k];
}
///////////////////////////////////////////////////

template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuEvalConv2D(KER Ker, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{
    // Data on the device.
    TYPE* x_d;
    TYPE* y_d;
    TYPE* beta_d;
    TYPE* gamma_d;
    TYPE* gammaB;

    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&y_d, sizeof(TYPE)*(ny*DIMPOINT));
    hipMalloc((void**)&beta_d, sizeof(TYPE)*(ny*DIMVECT));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMVECT));

    // Send data from host to device.
    hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y_h, sizeof(TYPE)*(ny*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(ny*DIMVECT), hipMemcpyHostToDevice);

    // Compute on device.
    dim3 blockSize;
    blockSize.x = 192; // number of threads in each block
    int blockSizey = blockSize.x;
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);
	gridSize.y =  ny / blockSizey + (ny%blockSizey==0 ? 0 : 1);

    hipMalloc((void**)&gammaB, sizeof(TYPE)*(nx*DIMVECT*gridSize.y));

    // Reduce  : grid and block are 1d
    dim3 blockSize2;
    blockSize2.x = 192; // number of threads in each block
    dim3 gridSize2;
    gridSize2.x =  (nx*DIMVECT) / blockSize2.x + ((nx*DIMVECT)%blockSize2.x==0 ? 0 : 1);

	GpuConv2DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
		<<<gridSize,blockSize,blockSize.x*(DIMVECT+DIMPOINT)*sizeof(TYPE)>>>
			(Ker, x_d, y_d, beta_d, gammaB, nx, ny);

    reduce0<TYPE,DIMVECT><<<gridSize2, blockSize2>>>(gammaB, gamma_d, gridSize.y,nx);

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMVECT),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(y_d);
    hipFree(beta_d);
    hipFree(gamma_d);
    hipFree(gammaB);

    return 0;
}



template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuEvalConv2D(TYPE sigma, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{

	return GpuEvalConv2D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
			x_h, y_h, beta_h, gamma_h, nx, ny);
}





////////////////////////////////////////
///// GRAD1 Conv2D ///////////////////////
////////////////////////////////////////


template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
__global__ void GpuGrad1Conv2DOnDevice(KER Ker,
        TYPE *alpha, TYPE *x, TYPE *y, TYPE *beta, TYPE *gammaB,
        int nx, int ny)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

    TYPE xi[DIMPOINT], alphai[DIMVECT], gammai[DIMPOINT];
    if(i<nx)  // we will compute gammai only if i is in the range
    {
        // load xi and alphai from device global memory
        for(int k=0; k<DIMPOINT; k++)
            xi[k] = x[i*DIMPOINT+k];
        for(int k=0; k<DIMVECT; k++)
            alphai[k] = alpha[i*DIMVECT+k];
        for(int k=0; k<DIMPOINT; k++)
            gammai[k] = 0.0f;
    }

        int j = blockIdx.y * blockDim.x + threadIdx.x;
        if(j<ny) // we load yj and betaj from device global memory only if j<ny
        {
            int inc = DIMPOINT + DIMVECT;
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+k] = y[j*DIMPOINT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
        }
        __syncthreads();
        if(i<nx) // we compute gammai only if i is in the range
        {
            TYPE *yj, *betaj;
            yj = SharedData;
            betaj = SharedData + DIMPOINT;
            int inc = DIMPOINT + DIMVECT;
            for(int jrel = 0; (jrel < blockDim.x) && ((blockDim.x*blockIdx.y+jrel)< ny); jrel++, yj+=inc, betaj+=inc)
	            Ker.Grad1(gammai,alphai,xi,yj,betaj);
        }
        __syncthreads();

    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMPOINT; k++)
            gammaB[blockIdx.y*DIMPOINT*nx+i*DIMPOINT+k] = gammai[k];
}

//////////////////////////////////////////////////////////////

template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuGrad1Conv2D(KER Ker, TYPE* alpha_h, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{

    // Data on the device.
    TYPE* x_d;
    TYPE* y_d;
    TYPE* alpha_d;
    TYPE* gamma_d;
    TYPE* gammaB;
    TYPE* beta_d;

    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&y_d, sizeof(TYPE)*(ny*DIMPOINT));
    hipMalloc((void**)&alpha_d, sizeof(TYPE)*(nx*DIMVECT));
    hipMalloc((void**)&beta_d, sizeof(TYPE)*(ny*DIMVECT));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMPOINT));

    // Send data from host to device.
    hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y_h, sizeof(TYPE)*(ny*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(alpha_d, alpha_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);
    hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(ny*DIMVECT), hipMemcpyHostToDevice);

    // compute on device.
    dim3 blockSize;
    blockSize.x = 192; // number of threads in each block
    int blockSizey = blockSize.x;
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);
    gridSize.y =  ny / blockSizey + (ny%blockSizey==0 ? 0 : 1);

    hipMalloc((void**)&gammaB, sizeof(TYPE)*(nx*DIMPOINT*gridSize.y));

   // Reduce  : grid and block are 1d
    dim3 blockSize2;
    blockSize2.x = 192; // number of threads in each block
    dim3 gridSize2;
    gridSize2.x =  (nx*DIMPOINT) / blockSize2.x + ((nx*DIMPOINT)%blockSize2.x==0 ? 0 : 1);

    GpuGrad1Conv2DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
		<<<gridSize,blockSize,blockSize.x*(DIMPOINT+DIMVECT)*sizeof(TYPE)>>>
			(Ker, alpha_d, x_d, y_d, beta_d, gammaB, nx, ny);

    reduce0<TYPE,DIMPOINT><<<gridSize2, blockSize2>>>(gammaB, gamma_d, gridSize.y,nx);

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMPOINT),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(y_d);
    hipFree(alpha_d);
    hipFree(gamma_d);
    hipFree(gammaB);
    hipFree(beta_d);

    return 0;
}


template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuGrad1Conv2D(TYPE sigma, TYPE* alpha_h, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{
	return GpuGrad1Conv2D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
			alpha_h, x_h, y_h, beta_h, gamma_h, nx, ny);
}


///////////////////////////////////////
////////// GRAD Conv2D //////////////////
///////////////////////////////////////


template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
__global__ void GpuGradConv2DOnDevice(KER Ker,
        TYPE *alpha, TYPE *x, TYPE *beta, TYPE *gammaB,
        int nx)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

    TYPE xi[DIMPOINT], alphai[DIMVECT], betai[DIMVECT], gammai[DIMPOINT];
    if(i<nx)  // we will compute gammai only if i is in the range
    {
        // load xi, alphai, betai from device global memory
        for(int k=0; k<DIMPOINT; k++)
            xi[k] = x[i*DIMPOINT+k];
        for(int k=0; k<DIMVECT; k++)
            alphai[k] = alpha[i*DIMVECT+k];
        for(int k=0; k<DIMVECT; k++)
            betai[k] = beta[i*DIMVECT+k];
        for(int k=0; k<DIMPOINT; k++)
            gammai[k] = 0.0f;
    }


        int j = blockIdx.y * blockDim.x + threadIdx.x;
        if(j<nx) // we load xj, alphaj and betaj from device global memory only if j<nx
        {
            int inc = DIMPOINT + 2 * DIMVECT;
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+k] = x[j*DIMPOINT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+k] = alpha[j*DIMVECT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+DIMVECT+k] = beta[j*DIMVECT+k];
        }
        __syncthreads();
        if(i<nx) // we compute gammai only if i is in the range
        {
            TYPE *xj, *alphaj, *betaj;
            xj = SharedData;
            alphaj = SharedData + DIMPOINT;
            betaj = SharedData + DIMPOINT + DIMVECT;
            int inc = DIMPOINT + 2 * DIMVECT;
            for(int jrel = 0; (jrel < blockDim.x) && ((blockDim.x*blockIdx.y+jrel)< nx); jrel++, xj+=inc, alphaj+=inc, betaj+=inc)
                Ker.Grad(gammai, xi, xj, alphai, alphaj, betai, betaj);
        }
        __syncthreads();


    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMPOINT; k++)
            gammaB[blockIdx.y*DIMPOINT*nx+i*DIMPOINT+k] = gammai[k];
}

////////////////////////////////////////////////////////////////////////////

template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuGradConv2D(KER Ker,
        TYPE* alpha_h, TYPE* x_h, TYPE* beta_h, TYPE* gamma_h,
         int nx)
{

    // Data on the device.
    TYPE* x_d;
    TYPE* alpha_d;
    TYPE* gamma_d;
    TYPE* gammaB;
    TYPE* beta_d;

    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&alpha_d, sizeof(TYPE)*(nx*DIMVECT));
    hipMalloc((void**)&beta_d, sizeof(TYPE)*(nx*DIMVECT));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMPOINT));

    // Send data from host to device.
    hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(alpha_d, alpha_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);
    hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);

    // compute on device.
    dim3 blockSize;
    blockSize.x = 192; // number of threads in each block
    int blockSizey = blockSize.x;
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);
    gridSize.y =  nx / blockSizey + (nx%blockSizey==0 ? 0 : 1);

    // Reduce  : grid and block are 1d
    dim3 blockSize2;
    blockSize2.x = 192; // number of threads in each block
    dim3 gridSize2;
    gridSize2.x =  (nx*DIMPOINT) / blockSize2.x + ((nx*DIMPOINT)%blockSize2.x==0 ? 0 : 1);

   hipMalloc((void**)&gammaB, sizeof(TYPE)*(nx*DIMPOINT*gridSize.y));

    GpuGradConv2DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
        <<<gridSize,blockSize,blockSize.x*(2*DIMPOINT+DIMVECT)*sizeof(TYPE)>>>
            (Ker, alpha_d, x_d, beta_d, gammaB, nx);

    reduce0<TYPE,DIMPOINT><<<gridSize2, blockSize2>>>(gammaB, gamma_d, gridSize.y,nx);

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMPOINT),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(alpha_d);
    hipFree(beta_d);
    hipFree(gamma_d);
    hipFree(gammaB);

    return 0;
}

template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuGradConv2D(TYPE sigma, TYPE* alpha_h, TYPE* x_h, TYPE* beta_h, TYPE* gamma_h, int nx)
{
	return GpuGradConv2D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
			alpha_h, x_h, beta_h, gamma_h, nx);
}

////////////////////////////////////////////
////////// GRAD DIFF Conv2D //////////////////
////////////////////////////////////////////


template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
__global__ void GpuGradDiffConv2DOnDevice(KER Ker,
        TYPE *x, TYPE *beta, TYPE *eta, TYPE *gammaB,
        int nx)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

    TYPE xi[DIMPOINT], betai[DIMVECT], etai[DIMPOINT], gammai[DIMPOINT];
    if(i<nx)  // we will compute gammai only if i is in the range
    {
        // load xi, etai, betai from device global memory
        for(int k=0; k<DIMPOINT; k++)
            xi[k] = x[i*DIMPOINT+k];
        for(int k=0; k<DIMVECT; k++)
            betai[k] = beta[i*DIMVECT+k];
        for(int k=0; k<DIMPOINT; k++)
            etai[k] = eta[i*DIMPOINT+k];
        for(int k=0; k<DIMPOINT; k++)
            gammai[k] = 0.0f;
    }


        int j = blockIdx.y * blockDim.x + threadIdx.x;
        if(j<nx) // we load xj, etaj and betaj from device global memory only if j<nx
        {
            int inc = 2 * DIMPOINT + DIMVECT;
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+k] = x[j*DIMPOINT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+DIMVECT+k] = eta[j*DIMPOINT+k];
        }
        __syncthreads();
        if(i<nx) // we compute gammai only if i is in the range
        {
            TYPE *xj, *betaj, *etaj;
            xj = SharedData;
            betaj = SharedData + DIMPOINT;
            etaj = SharedData + DIMPOINT + DIMVECT;
            int inc = 2 * DIMPOINT + DIMVECT;
            for(int jrel = 0; (jrel < blockDim.x) && ((blockDim.x*blockIdx.y+jrel)< nx); jrel++, xj+=inc, betaj+=inc, etaj+=inc)
                Ker.GradDiff(gammai, xi, xj, betai, betaj, etai, etaj);
        }
        __syncthreads();


    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMPOINT; k++)
            gammaB[blockIdx.y*DIMPOINT*nx+i*DIMPOINT+k] = gammai[k];
}

////////////////////////////////////////////////////////////////////////////

template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuGradDiffConv2D(KER Ker,
        TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h,
         int nx)
{

    // Data on the device.
    TYPE* x_d;
    TYPE* beta_d;
    TYPE* gamma_d;
    TYPE* gammaB;
    TYPE* eta_d;

    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&beta_d, sizeof(TYPE)*(nx*DIMVECT));
    hipMalloc((void**)&eta_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMPOINT));

    // Send data from host to device.
    hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);
    hipMemcpy(eta_d, eta_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);

    // compute on device.
    dim3 blockSize;
    blockSize.x = 192; // number of threads in each block
    int blockSizey = blockSize.x;
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);
    gridSize.y =  nx / blockSizey + (nx%blockSizey==0 ? 0 : 1);

    hipMalloc((void**)&gammaB, sizeof(TYPE)*(nx*DIMPOINT*gridSize.y));

    // Reduce  : grid and block are 1d
    dim3 blockSize2;
    blockSize2.x = 192; // number of threads in each block
    dim3 gridSize2;
    gridSize2.x =  (nx*DIMPOINT) / blockSize2.x + ((nx*DIMPOINT)%blockSize2.x==0 ? 0 : 1);

    GpuGradDiffConv2DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
        <<<gridSize,blockSize,blockSize.x*(2*DIMPOINT+DIMVECT)*sizeof(TYPE)>>>
            (Ker, x_d, beta_d, eta_d, gammaB, nx);

    reduce0<TYPE,DIMPOINT><<<gridSize2, blockSize2>>>(gammaB, gamma_d, gridSize.y,nx);

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMPOINT),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(eta_d);
    hipFree(beta_d);
    hipFree(gamma_d);
    hipFree(gammaB);

    return 0;
}

template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuGradDiffConv2D(TYPE sigma, TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h, int nx)
{
	return GpuGradDiffConv2D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
			x_h, beta_h, eta_h, gamma_h, nx);
}


////////////////////////////////////////////
////////// DIFF Conv2D ///////////////////////
////////////////////////////////////////////


template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
__global__ void GpuDiffConv2DOnDevice(KER Ker,
        TYPE *x, TYPE *beta, TYPE *eta, TYPE *gammaB,
        int nx)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

    TYPE xi[DIMPOINT], etai[DIMPOINT], gammai[DIMVECT];
    if(i<nx)  // we will compute gammai only if i is in the range
    {
        // load xi, etai from device global memory
        for(int k=0; k<DIMPOINT; k++)
            xi[k] = x[i*DIMPOINT+k];
        for(int k=0; k<DIMPOINT; k++)
            etai[k] = eta[i*DIMPOINT+k];
        for(int k=0; k<DIMVECT; k++)
            gammai[k] = 0.0f;
    }


        int j = blockIdx.y * blockDim.x + threadIdx.x;
        if(j<nx) // we load xj, betaj and etaj from device global memory only if j<nx
        {
            int inc = 2 * DIMPOINT + DIMVECT;
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+k] = x[j*DIMPOINT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+DIMVECT+k] = eta[j*DIMPOINT+k];
        }
        __syncthreads();
        if(i<nx) // we compute gammai only if i is in the range
        {
            TYPE *xj, *betaj, *etaj;
            xj = SharedData;
            betaj = SharedData + DIMPOINT;
            etaj = SharedData + DIMPOINT + DIMVECT;
            int inc = 2 * DIMPOINT + DIMVECT;
            for(int jrel = 0; (jrel < blockDim.x) && ((blockDim.x*blockIdx.y+jrel)< nx); jrel++, xj+=inc, betaj+=inc, etaj+=inc)
                Ker.Diff(gammai, xi, xj, betaj, etai, etaj);
        }
        __syncthreads();


    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMVECT; k++)
            gammaB[blockIdx.y*DIMVECT*nx+i*DIMVECT+k] = gammai[k];
}

////////////////////////////////////////////////////////////////////////////

template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuDiffConv2D(KER Ker,
        TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h,
        int nx)
{

    // Data on the device.
    TYPE* x_d;
    TYPE* beta_d;
    TYPE* gamma_d;
    TYPE* gammaB;
    TYPE* eta_d;

    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&beta_d, sizeof(TYPE)*(nx*DIMVECT));
    hipMalloc((void**)&eta_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMVECT));

    // Send data from host to device.
    hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);
    hipMemcpy(eta_d, eta_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);

    // compute on device.
    dim3 blockSize;
    blockSize.x = 192; // number of threads in each block
    int blockSizey = blockSize.x;
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);
    gridSize.y =  nx / blockSizey + (nx%blockSizey==0 ? 0 : 1);

    hipMalloc((void**)&gammaB, sizeof(TYPE)*(nx*DIMVECT*gridSize.y));

    // Reduce  : grid and block are 1d
    dim3 blockSize2;
    blockSize2.x = 192; // number of threads in each block
    dim3 gridSize2;
    gridSize2.x =  (nx*DIMVECT) / blockSize2.x + ((nx*DIMVECT)%blockSize2.x==0 ? 0 : 1);

    GpuDiffConv2DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
        <<<gridSize,blockSize,blockSize.x*(2*DIMPOINT+DIMVECT)*sizeof(TYPE)>>>
            (Ker, x_d, beta_d, eta_d, gammaB, nx);

    reduce0<TYPE,DIMVECT><<<gridSize2, blockSize2>>>(gammaB, gamma_d, gridSize.y,nx);

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMVECT),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(eta_d);
    hipFree(beta_d);
    hipFree(gamma_d);
    hipFree(gammaB);

    return 0;
}

template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuDiffConv2D(TYPE sigma, TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h, int nx)
{
	return GpuDiffConv2D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
			x_h, beta_h, eta_h, gamma_h, nx);
}




// http://www.parashift.com/c++-faq-lite/separate-template-fn-defn-from-decl.html
#define DECLARE_Conv2DS(TYPE,DIMPOINT,DIMVECT) \
	template int GaussGpuEvalConv2D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int, int); \
	template int GaussGpuGrad1Conv2D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, TYPE*, int, int); \
	template int GaussGpuGradConv2D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int); \
	template int GaussGpuGradDiffConv2D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int); \
	template int GaussGpuDiffConv2D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int);
#define DECLARE_Conv2DS_ALLDIMS_FOR(TYPE) \
	DECLARE_Conv2DS(TYPE,1,1) \
	DECLARE_Conv2DS(TYPE,2,1) \
	DECLARE_Conv2DS(TYPE,2,2) \
	DECLARE_Conv2DS(TYPE,2,4) \
	DECLARE_Conv2DS(TYPE,3,1) \
	DECLARE_Conv2DS(TYPE,3,3) \
	DECLARE_Conv2DS(TYPE,3,6)
DECLARE_Conv2DS_ALLDIMS_FOR(float)
DECLARE_Conv2DS_ALLDIMS_FOR(double)




#endif /* _GpuConv2D_cu */
