#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <assert.h>
#include <time.h>

#include "GpuConv1D.cu"
#include "GpuConv2D.cu"


#define DIMPOINT 2 // dimension of the ambiant space : for curve ==2 or 3; for surface ==3
#define DIMVECT 2 // dimension of the object : for curve ==2; for surface ==3


// Compilation instruction : nvcc -arch=sm_?? conv.cu -o conv


/////////////////////////////////////
//               MAIN
/////////////////////////////////////


int main()
{

	/*---------*/
	/*  init   */
	/*---------*/

	float* x_h;
	float* y_h;
	float* beta_h;

	// Dimension of the problem 
	int nx = 100; // nbr of "row" (== nbre of point in the first object) may vary from 100 to 100000
	int ny = 1000000; // nbr of "column" (== nbre of point in the second object) may vary from 2000 to 100000 

	// arbitrary data
	x_h = (float *) malloc (nx * DIMPOINT * sizeof(float));
	for (int i=0;i<nx*DIMPOINT;i++){x_h[i] = (float)i/((float)nx*3.0f);}

	y_h = (float *) malloc (ny * DIMPOINT * sizeof(float));
	for (int i=0;i<ny*DIMPOINT;i++){y_h[i] = logf( ((float)i+1.0f)/((float)ny*3.0f));}
	
	beta_h = (float *) malloc (ny * DIMVECT * sizeof(float));
	for (int i=0;i<ny*DIMVECT;i++){beta_h[i] = cosf((float)i*(float)i/((float)ny*3.0f));}


	/*----------*/
	/*  kernel  */
	/*----------*/

	printf("\n");

	///////////
	// Conv2 //
	///////////

	clock_t tic2 = clock();
	// pointer to output
	float* gamma_h2;
	gamma_h2 = (float *) malloc (nx * DIMVECT * sizeof(float));	
	GaussGpuEvalConv2D<float,DIMPOINT,DIMVECT>(0.5,x_h, y_h, beta_h, gamma_h2, nx, ny);


	clock_t toc2 = clock();
	printf("Conv2d with nx=%d and ny=%d took %f seconds\n",nx,ny ,(double)(toc2 - tic2) / CLOCKS_PER_SEC);

	///////////////
	// Conv2 bis //
	///////////////

	clock_t tic2_bis = clock();
	// pointer to output
	float* gamma_h2_bis;
	gamma_h2_bis = (float *) malloc (nx * DIMVECT * sizeof(float));	
	GaussGpuEvalConv2D<float,DIMPOINT,DIMVECT>(0.5,x_h, y_h, beta_h, gamma_h2_bis, nx, ny);
	clock_t toc2_bis = clock();
	printf("Conv2d bis with nx=%d and ny=%d took %f seconds\n",nx,ny ,(double)(toc2_bis - tic2_bis) / CLOCKS_PER_SEC);

	///////////////
	// Conv2 ter //
	///////////////

	clock_t tic2_ter = clock();
	// pointer to output
	float* gamma_h2_ter;
	gamma_h2_ter = (float *) malloc (nx * DIMVECT * sizeof(float));	
	GaussGpuEvalConv2D<float,DIMPOINT,DIMVECT>(0.5,x_h, y_h, beta_h, gamma_h2_ter, nx, ny);
	clock_t toc2_ter = clock();
	printf("Conv2d ter with nx=%d and ny=%d took %f seconds\n",nx,ny ,(double)(toc2_ter - tic2_ter) / CLOCKS_PER_SEC);

	///////////
	// Conv1 //
	///////////

	clock_t tic1 = clock();
	// pointer to output
	float* gamma_h1;
	gamma_h1 = (float *) malloc (nx * DIMVECT * sizeof(float));
	GaussGpuEvalConv1D<float,DIMPOINT,DIMVECT>(0.5,x_h, y_h, beta_h, gamma_h1, nx, ny);
	clock_t toc1 = clock();
	printf("Conv1d with nx=%d and ny=%d took %f seconds\n",nx,ny ,(double)(toc1 - tic1) / CLOCKS_PER_SEC);

	///////////////
	// Conv1 bis //
	///////////////

	clock_t tic1_bis = clock();
	// pointer to output
	float* gamma_h1_bis;
	gamma_h1_bis = (float *) malloc (nx * DIMVECT * sizeof(float));	
	GaussGpuEvalConv1D<float,DIMPOINT,DIMVECT>(0.5,x_h, y_h, beta_h, gamma_h1_bis, nx, ny);
	clock_t toc1_bis = clock();
	printf("Conv1d bis with nx=%d and ny=%d took %f seconds\n",nx,ny ,(double)(toc1_bis - tic1_bis) / CLOCKS_PER_SEC);

	///////////////
	// Conv1 ter //
	///////////////

	clock_t tic1_ter = clock();
	// pointer to output
	float* gamma_h1_ter;
	gamma_h1_ter = (float *) malloc (nx * DIMVECT * sizeof(float));	
	GaussGpuEvalConv1D<float,DIMPOINT,DIMVECT>(0.5,x_h, y_h, beta_h, gamma_h1_ter, nx, ny);
	clock_t toc1_ter = clock();
	printf("Conv1d ter with nx=%d and ny=%d took %f seconds\n",nx,ny ,(double)(toc1_ter - tic1_ter) / CLOCKS_PER_SEC);


	/*--------*/
	/* Output */
	/*--------*/

	float err =0;
	for (int i=0;i<DIMVECT*nx;i++){
	       float erR = fabs((gamma_h1_bis[i] - gamma_h2_bis[i]) / gamma_h1_bis[i]) ;
	       if (erR > err) {
	          err = erR;
	       }		  
	}

	printf("Max relative error : %f \n\n",err);
	return 0;

}



