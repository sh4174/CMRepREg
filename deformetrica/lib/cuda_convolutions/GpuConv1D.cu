#include "hip/hip_runtime.h"
/***************************************************************************************
*                                                                                      *
*                                     Deformetrica                                     *
*                                                                                      *
*    Copyright Inria and the University of Utah.  All rights reserved. This file is    *
*    distributed under the terms of the MIT License. This file is also distributed     *
*    under the terms of the Inria Non-Commercial License Agreement.                    *
*                                                                                      *
*                                                                                      *
****************************************************************************************/

#ifndef _GpuConv1D_cu
#define _GpuConv1D_cu

#include <stdio.h>

//#include "CauchyFunction.h"
#include "GaussFunction.h"
#include "ScalarRadialKernel.h"

#include "GpuConv1D.h"



////////////////////////////////////////////////////////////////////////////////////////////////////
// Conv1D :
////////////////////////////////////////////////////////////////////////////////////////////////////

// Thread kernel: computation of \f$ \gamma_i = \sum_j K(x_i,y_j)\beta_j for index i given by thread id.
template < typename TYPE, int DIMPOINT, int DIMVECT, class KER  >
__global__ void GpuConv1DOnDevice(KER Ker,
                                      TYPE *x, TYPE *y, TYPE *beta, TYPE *gamma,
                                      int nx, int ny)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

    TYPE xi[DIMPOINT], gammai[DIMVECT];
    if(i<nx)  // we will compute gammai only if i is in the range
    {
        // load xi from device global memory
        for(int k=0; k<DIMPOINT; k++)
            xi[k] = x[i*DIMPOINT+k];
        for(int k=0; k<DIMVECT; k++)
            gammai[k] = 0.0f;
    }

    for(int jstart = 0, tile = 0; jstart < ny; jstart += blockDim.x, tile++)
    {
        int j = tile * blockDim.x + threadIdx.x;
        if(j<ny) // we load yj and betaj from device global memory only if j<ny
        {
            int inc = DIMPOINT + DIMVECT;
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+k] = y[j*DIMPOINT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
        }
        __syncthreads();
        
        if(i<nx) // we compute gammai only if needed
        {
            TYPE *yj, *betaj;
            yj = SharedData;
            betaj = SharedData + DIMPOINT;
            int inc = DIMPOINT + DIMVECT;
            for(int jrel = 0; jrel < blockDim.x && jrel<ny-jstart; jrel++, yj+=inc, betaj+=inc)
		Ker.Eval(gammai,xi,yj,betaj);
    	}
	__syncthreads();
    }

    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMVECT; k++)
            gamma[i*DIMVECT+k] = gammai[k];
}



template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuEvalConv1D(KER Ker, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{
    // Data on the device.
    TYPE* x_d;
    TYPE* y_d;
    TYPE* beta_d;
    TYPE* gamma_d;

    /*
	printf("x_h(cuda) = \n");
	for(unsigned int index = 0; index < (DIMPOINT*nx); index++)
	{
		printf("   %f", x_h[index]);
		if(index%DIMVECT-2==0)
			printf("\n");
	}
	printf("y_h(cuda) = \n");
	for(unsigned int index = 0; index < (DIMPOINT*ny); index++)
	{
		printf("   %f", y_h[index]);
		if(index%DIMVECT-2==0)
			printf("\n");
	}
	printf("beta_h(cuda) = \n");
	for(unsigned int index = 0; index < (DIMPOINT*ny); index++)
	{
		printf("   %f", beta_h[index]);
		if(index%DIMVECT-2==0)
			printf("\n");
	}
	*/


    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&y_d, sizeof(TYPE)*(ny*DIMPOINT));
    hipMalloc((void**)&beta_d, sizeof(TYPE)*(ny*DIMVECT));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMVECT));

    // Send data from host to device.
    hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y_h, sizeof(TYPE)*(ny*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(ny*DIMVECT), hipMemcpyHostToDevice);

    // Compute on device.
    dim3 blockSize;
    blockSize.x = 192; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);
	
	GpuConv1DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
		<<<gridSize,blockSize,blockSize.x*(DIMVECT+DIMPOINT)*sizeof(TYPE)>>>
			(Ker, x_d, y_d, beta_d, gamma_d, nx, ny);

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMVECT),hipMemcpyDeviceToHost);

/*    hipDeviceSynchronize();
	printf("gamma_h(cuda) = \n");
	for(unsigned int index = 0; index < (DIMPOINT*nx); index++)
	{
		printf("   %f", gamma_h[index]);
		if(index%DIMVECT-2==0)
			printf("\n");
	}
	*/
    // Free memory.
    hipFree(x_d);
    hipFree(y_d);
    hipFree(beta_d);
    hipFree(gamma_d);

    return 0;
}


/*
template < typename TYPE, int DIMPOINT, int DIMVECT >
int CauchyGpuEvalConv1D(TYPE sigma, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{

	return GpuEvalConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> >(CauchyFunction<TYPE>(sigma)),
			x_h, y_h, beta_h, gamma_h, nx, ny);
}
*/


template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuEvalConv1D(TYPE sigma, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{
	
	return GpuEvalConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
			x_h, y_h, beta_h, gamma_h, nx, ny);
}





////////////////////////////////////////////////////////////////////////////////////////////////////
// Grad1 Conv1D :
////////////////////////////////////////////////////////////////////////////////////////////////////

template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
__global__ void GpuGrad1Conv1DOnDevice(KER Ker,
        TYPE *alpha, TYPE *x, TYPE *y, TYPE *beta, TYPE *gamma,
        int nx, int ny)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

    TYPE xi[DIMPOINT], alphai[DIMVECT], gammai[DIMPOINT];
    if(i<nx)  // we will compute gammai only if i is in the range
    {
        // load xi and alphai from device global memory
        for(int k=0; k<DIMPOINT; k++)
            xi[k] = x[i*DIMPOINT+k];
        for(int k=0; k<DIMVECT; k++)
            alphai[k] = alpha[i*DIMVECT+k];
        for(int k=0; k<DIMPOINT; k++)
            gammai[k] = 0.0f;
    }

    for(int jstart = 0, tile = 0; jstart < ny; jstart += blockDim.x, tile++)
    {
        int j = tile * blockDim.x + threadIdx.x;
        if(j<ny) // we load yj and betaj from device global memory only if j<ny
        {
            int inc = DIMPOINT + DIMVECT;
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+k] = y[j*DIMPOINT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
        }
        __syncthreads();
        if(i<nx) // we compute gammai only if i is in the range
        {
            TYPE *yj, *betaj;
            yj = SharedData;
            betaj = SharedData + DIMPOINT;
            int inc = DIMPOINT + DIMVECT;
            for(int jrel = 0; jrel < blockDim.x && jrel<ny-jstart; jrel++, yj+=inc, betaj+=inc)
	            Ker.Grad1(gammai,alphai,xi,yj,betaj);
        }
        __syncthreads();
    }

    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMPOINT; k++)
            gamma[i*DIMPOINT+k] = gammai[k];
}



template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuGrad1Conv1D(KER Ker, TYPE* alpha_h, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{

    // Data on the device.
    TYPE* x_d;
    TYPE* y_d;
    TYPE* alpha_d;
    TYPE* gamma_d;
    TYPE* beta_d;

    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&y_d, sizeof(TYPE)*(ny*DIMPOINT));
    hipMalloc((void**)&alpha_d, sizeof(TYPE)*(nx*DIMVECT));
    hipMalloc((void**)&beta_d, sizeof(TYPE)*(ny*DIMVECT));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMPOINT));

    // Send data from host to device.
    hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y_h, sizeof(TYPE)*(ny*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(alpha_d, alpha_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);
    hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(ny*DIMVECT), hipMemcpyHostToDevice);

    // compute on device.
    dim3 blockSize;
    blockSize.x = 192; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

    GpuGrad1Conv1DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
		<<<gridSize,blockSize,blockSize.x*(DIMPOINT+DIMVECT)*sizeof(TYPE)>>>
			(Ker, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMPOINT),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(y_d);
    hipFree(alpha_d);
    hipFree(gamma_d);
    hipFree(beta_d);

    return 0;
}


/*
template < typename TYPE, int DIMPOINT, int DIMVECT >
int CauchyGpuGrad1Conv1D(TYPE sigma, TYPE* alpha_h, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{
	return GpuGrad1Conv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> >(CauchyFunction<TYPE>(sigma)),
			alpha_h, x_h, y_h, beta_h, gamma_h, nx, ny);
}
*/



template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuGrad1Conv1D(TYPE sigma, TYPE* alpha_h, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{
	return GpuGrad1Conv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
			alpha_h, x_h, y_h, beta_h, gamma_h, nx, ny);
}



//
//
//////////////////////////////////////////////////////////////////////////////////////////////////////
//// Grad Conv1D :
//////////////////////////////////////////////////////////////////////////////////////////////////////
//
//template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
//__global__ void GpuGradConv1DOnDevice(KER Ker,
//        TYPE *alpha, TYPE *x, TYPE *beta, TYPE *gamma,
//        int nx)
//{
//    int i = blockIdx.x * blockDim.x + threadIdx.x;
//
//    extern __shared__ char SharedData_char[];
//    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);
//
//    TYPE xi[DIMPOINT], alphai[DIMVECT], betai[DIMVECT], gammai[DIMPOINT];
//    if(i<nx)  // we will compute gammai only if i is in the range
//    {
//        // load xi, alphai, betai from device global memory
//        for(int k=0; k<DIMPOINT; k++)
//            xi[k] = x[i*DIMPOINT+k];
//        for(int k=0; k<DIMVECT; k++)
//            alphai[k] = alpha[i*DIMVECT+k];
//        for(int k=0; k<DIMVECT; k++)
//            betai[k] = beta[i*DIMVECT+k];
//        for(int k=0; k<DIMPOINT; k++)
//            gammai[k] = 0.0f;
//    }
//
//    for(int jstart = 0, tile = 0; jstart < nx; jstart += blockDim.x, tile++)
//    {
//        int j = tile * blockDim.x + threadIdx.x;
//        if(j<nx) // we load xj, alphaj and betaj from device global memory only if j<nx
//        {
//            int inc = DIMPOINT + 2 * DIMVECT;
//            for(int k=0; k<DIMPOINT; k++)
//                SharedData[threadIdx.x*inc+k] = x[j*DIMPOINT+k];
//            for(int k=0; k<DIMVECT; k++)
//                SharedData[threadIdx.x*inc+DIMPOINT+k] = alpha[j*DIMVECT+k];
//            for(int k=0; k<DIMVECT; k++)
//                SharedData[threadIdx.x*inc+DIMPOINT+DIMVECT+k] = beta[j*DIMVECT+k];
//        }
//        __syncthreads();
//        if(i<nx) // we compute gammai only if i is in the range
//        {
//            TYPE *xj, *alphaj, *betaj;
//            xj = SharedData;
//            alphaj = SharedData + DIMPOINT;
//            betaj = SharedData + DIMPOINT + DIMVECT;
//            int inc = DIMPOINT + 2 * DIMVECT;
//            for(int jrel = 0; jrel < blockDim.x && jrel<nx-jstart; jrel++, xj+=inc, alphaj+=inc, betaj+=inc)
//                Ker.Grad(gammai, xi, xj, alphai, alphaj, betai, betaj);
//        }
//        __syncthreads();
//    }
//
//    // Save the result in global memory.
//    if(i<nx)
//        for(int k=0; k<DIMPOINT; k++)
//            gamma[i*DIMPOINT+k] = gammai[k];
//}
//
//
//
//template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
//int GpuGradConv1D(KER Ker,
//        TYPE* alpha_h, TYPE* x_h, TYPE* beta_h, TYPE* gamma_h,
//         int nx)
//{
//
//    // Data on the device.
//    TYPE* x_d;
//    TYPE* alpha_d;
//    TYPE* gamma_d;
//    TYPE* beta_d;
//
//    // Allocate arrays on device.
//    hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
//    hipMalloc((void**)&alpha_d, sizeof(TYPE)*(nx*DIMVECT));
//    hipMalloc((void**)&beta_d, sizeof(TYPE)*(nx*DIMVECT));
//    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMPOINT));
//
//    // Send data from host to device.
//    hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
//    hipMemcpy(alpha_d, alpha_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);
//    hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);
//
//    // compute on device.
//    dim3 blockSize;
//    blockSize.x = 192; // number of threads in each block
//    dim3 gridSize;
//    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);
//
//	GpuGradConv1DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
//        <<<gridSize,blockSize,blockSize.x*(2*DIMPOINT+DIMVECT)*sizeof(TYPE)>>>
//            (Ker, alpha_d, x_d, beta_d, gamma_d, nx);
//
//    // block until the device has completed
//    hipDeviceSynchronize();
//
//    // Send data from device to host.
//    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMPOINT),hipMemcpyDeviceToHost);
//
//    // Free memory.
//    hipFree(x_d);
//    hipFree(alpha_d);
//    hipFree(beta_d);
//    hipFree(gamma_d);
//
//    return 0;
//}
//
//
//
//template < typename TYPE, int DIMPOINT, int DIMVECT >
//int CauchyGpuGradConv1D(TYPE sigma, TYPE* alpha_h, TYPE* x_h, TYPE* beta_h, TYPE* gamma_h, int nx)
//{
//	return GpuGradConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> > >
//		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> >(CauchyFunction<TYPE>(sigma)),
//			alpha_h, x_h, beta_h, gamma_h, nx);
//}
//
//
//
//template < typename TYPE, int DIMPOINT, int DIMVECT >
//int GaussGpuGradConv1D(TYPE sigma, TYPE* alpha_h, TYPE* x_h, TYPE* beta_h, TYPE* gamma_h, int nx)
//{
//	return GpuGradConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
//		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
//			alpha_h, x_h, beta_h, gamma_h, nx);
//}
//




////////////////////////////////////////////////////////////////////////////////////////////////////
// GradDiff Conv1D :
////////////////////////////////////////////////////////////////////////////////////////////////////

template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
__global__ void GpuGradDiffConv1DOnDevice(KER Ker,
        TYPE *x, TYPE *beta, TYPE *eta, TYPE *gamma,
        int nx)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

    TYPE xi[DIMPOINT], betai[DIMVECT], etai[DIMPOINT], gammai[DIMPOINT];
    if(i<nx)  // we will compute gammai only if i is in the range
    {
        // load xi, etai, betai from device global memory
        for(int k=0; k<DIMPOINT; k++)
            xi[k] = x[i*DIMPOINT+k];
        for(int k=0; k<DIMVECT; k++)
            betai[k] = beta[i*DIMVECT+k];
        for(int k=0; k<DIMPOINT; k++)
            etai[k] = eta[i*DIMPOINT+k];
        for(int k=0; k<DIMPOINT; k++)
            gammai[k] = 0.0f;
    }

    for(int jstart = 0, tile = 0; jstart < nx; jstart += blockDim.x, tile++)
    {
        int j = tile * blockDim.x + threadIdx.x;
        if(j<nx) // we load xj, etaj and betaj from device global memory only if j<nx
        {
            int inc = 2 * DIMPOINT + DIMVECT;
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+k] = x[j*DIMPOINT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+DIMVECT+k] = eta[j*DIMPOINT+k];
        }
        __syncthreads();
        if(i<nx) // we compute gammai only if i is in the range
        {
            TYPE *xj, *betaj, *etaj;
            xj = SharedData;
            betaj = SharedData + DIMPOINT;
            etaj = SharedData + DIMPOINT + DIMVECT;
            int inc = 2 * DIMPOINT + DIMVECT;
            for(int jrel = 0; jrel < blockDim.x && jrel<nx-jstart; jrel++, xj+=inc, betaj+=inc, etaj+=inc)
                Ker.GradDiff(gammai, xi, xj, betai, betaj, etai, etaj);
        }
        __syncthreads();
    }

    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMPOINT; k++)
            gamma[i*DIMPOINT+k] = gammai[k];
}



template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuGradDiffConv1D(KER Ker,
        TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h,
         int nx)
{

    // Data on the device.
    TYPE* x_d;
    TYPE* beta_d;
    TYPE* gamma_d;
    TYPE* eta_d;

    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&beta_d, sizeof(TYPE)*(nx*DIMVECT));
    hipMalloc((void**)&eta_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMPOINT));

    // Send data from host to device.
    hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);
    hipMemcpy(eta_d, eta_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);

    // compute on device.
    dim3 blockSize;
    blockSize.x = 192; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

    GpuGradDiffConv1DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
        <<<gridSize,blockSize,blockSize.x*(2*DIMPOINT+DIMVECT)*sizeof(TYPE)>>>
            (Ker, x_d, beta_d, eta_d, gamma_d, nx);

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMPOINT),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(eta_d);
    hipFree(beta_d);
    hipFree(gamma_d);

    return 0;
}


/*
template < typename TYPE, int DIMPOINT, int DIMVECT >
int CauchyGpuGradDiffConv1D(TYPE sigma, TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h, int nx)
{
	return GpuGradDiffConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> >(CauchyFunction<TYPE>(sigma)),
			x_h, beta_h, eta_h, gamma_h, nx);
}
*/


template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuGradDiffConv1D(TYPE sigma, TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h, int nx)
{
	return GpuGradDiffConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
			x_h, beta_h, eta_h, gamma_h, nx);
}





////////////////////////////////////////////////////////////////////////////////////////////////////
// Diff Conv1D :
////////////////////////////////////////////////////////////////////////////////////////////////////

template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
__global__ void GpuDiffConv1DOnDevice(KER Ker,
        TYPE *x, TYPE *beta, TYPE *eta, TYPE *gamma,
        int nx)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

    TYPE xi[DIMPOINT], etai[DIMPOINT], gammai[DIMVECT];
    if(i<nx)  // we will compute gammai only if i is in the range
    {
        // load xi, etai from device global memory
        for(int k=0; k<DIMPOINT; k++)
            xi[k] = x[i*DIMPOINT+k];
        for(int k=0; k<DIMPOINT; k++)
            etai[k] = eta[i*DIMPOINT+k];
        for(int k=0; k<DIMVECT; k++)
            gammai[k] = 0.0f;
    }

    for(int jstart = 0, tile = 0; jstart < nx; jstart += blockDim.x, tile++)
    {
        int j = tile * blockDim.x + threadIdx.x;
        if(j<nx) // we load xj, betaj and etaj from device global memory only if j<nx
        {
            int inc = 2 * DIMPOINT + DIMVECT;
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+k] = x[j*DIMPOINT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+DIMVECT+k] = eta[j*DIMPOINT+k];
        }
        __syncthreads();
        if(i<nx) // we compute gammai only if i is in the range
        {
            TYPE *xj, *betaj, *etaj;
            xj = SharedData;
            betaj = SharedData + DIMPOINT;
            etaj = SharedData + DIMPOINT + DIMVECT;
            int inc = 2 * DIMPOINT + DIMVECT;
            for(int jrel = 0; jrel < blockDim.x && jrel<nx-jstart; jrel++, xj+=inc, betaj+=inc, etaj+=inc)
                Ker.Diff(gammai, xi, xj, betaj, etai, etaj);
        }
        __syncthreads();
    }

    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMVECT; k++)
            gamma[i*DIMVECT+k] = gammai[k];
}



template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuDiffConv1D(KER Ker,
        TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h,
        int nx)
{

    // Data on the device.
    TYPE* x_d;
    TYPE* beta_d;
    TYPE* gamma_d;
    TYPE* eta_d;

    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&beta_d, sizeof(TYPE)*(nx*DIMVECT));
    hipMalloc((void**)&eta_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMVECT));

    // Send data from host to device.
    hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);
    hipMemcpy(eta_d, eta_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);

    // compute on device.
    dim3 blockSize;
    blockSize.x = 192; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

    GpuDiffConv1DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
        <<<gridSize,blockSize,blockSize.x*(2*DIMPOINT+DIMVECT)*sizeof(TYPE)>>>
            (Ker, x_d, beta_d, eta_d, gamma_d, nx);

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMVECT),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(eta_d);
    hipFree(beta_d);
    hipFree(gamma_d);

    return 0;
}


/*
template < typename TYPE, int DIMPOINT, int DIMVECT >
int CauchyGpuDiffConv1D(TYPE sigma, TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h, int nx)
{
	return GpuDiffConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> >(CauchyFunction<TYPE>(sigma)),
			x_h, beta_h, eta_h, gamma_h, nx);
}
*/


template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuDiffConv1D(TYPE sigma, TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h, int nx)
{
	return GpuDiffConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)), 
			x_h, beta_h, eta_h, gamma_h, nx);
}





// http://www.parashift.com/c++-faq-lite/separate-template-fn-defn-from-decl.html
#define DECLARE_Conv1DS(TYPE,DIMPOINT,DIMVECT) \
	template int GaussGpuEvalConv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int, int); \
	template int GaussGpuGrad1Conv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, TYPE*, int, int); \
	template int GaussGpuGradDiffConv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int); \
	template int GaussGpuDiffConv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int);
#define DECLARE_Conv1DS_ALLDIMS_FOR(TYPE) \
	DECLARE_Conv1DS(TYPE,1,1) \
	DECLARE_Conv1DS(TYPE,2,1) \
	DECLARE_Conv1DS(TYPE,2,2) \
	DECLARE_Conv1DS(TYPE,2,4) \
	DECLARE_Conv1DS(TYPE,3,1) \
	DECLARE_Conv1DS(TYPE,3,3) \
	DECLARE_Conv1DS(TYPE,3,6)
DECLARE_Conv1DS_ALLDIMS_FOR(float)
DECLARE_Conv1DS_ALLDIMS_FOR(double)




#endif /* _GpuConv1D_cu */
